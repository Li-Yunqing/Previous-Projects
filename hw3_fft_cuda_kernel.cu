#include "hip/hip_runtime.h"
﻿#include<hip/hip_complex.h>

#include<iostream>

#include<math.h>

#include<hip/hip_math_constants.h>

#include<stdio.h>

#include <time.h>



using namespace std;



__host__ __device__ hipDoubleComplex complexp(double exp) {

	double a = cos(exp);

	double bi = sin(exp);

	return make_hipDoubleComplex(a, bi);

}



__global__ void fft(hipDoubleComplex* A, long int m) {


	unsigned int th = blockIdx.x * blockDim.x + threadIdx.x;

	unsigned int k = th / (m / 2);

	unsigned int j = th % (m / 2);


	hipDoubleComplex w = complexp(((2 * HIP_PI) / m) * j);

	hipDoubleComplex t = hipCmul(w, A[k + j + m / 2]);

	hipDoubleComplex u = A[k + j];

	A[k + j] = hipCadd(u, t);

	A[k + j + m / 2] = hipCsub(u, t);

}



__global__ void bit_reverse_copy(hipDoubleComplex* A, long int size, hipDoubleComplex* R) {

	int n = blockIdx.x * blockDim.x + threadIdx.x;

	if (n > size) return;

	int s = (int)log2((double)size);

	int revn = 0;

	for (int i = 0; i < s; i++) {

		revn += ((n >> i) & 1) << ((s - 1) - i);

	}

	hipDoubleComplex aux = A[n];

	R[revn] = aux;

}



int main() {

	int p;

	cin >> p;



	long int n = (long int)pow(2, p);

	size_t size = n * sizeof(hipDoubleComplex);

	hipDoubleComplex* A = (hipDoubleComplex*)malloc(size);





	for (long int k = 0; k < n; k++) {

		/*A[k].x = sin(k * (2 * HIP_PI_HI / n) );

		A[k].y = 0;*/

		if (k < n / 2) {

			A[k].x = 1;

			A[k].y = 0;

		}
		else {

			A[k].x = 0;

			A[k].y = 0;

		}

	}


	hipEvent_t stt, stp, stt2, stp2;

	hipEventCreate(&stt);

	hipEventCreate(&stp);

	hipEventCreate(&stt2);

	hipEventCreate(&stp2);


	hipEventRecord(stt);



	hipDoubleComplex* A_d, * B_d;

	hipMalloc(&A_d, size);

	hipMalloc(&B_d, size);

	hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);


	hipEventRecord(stt2);


	unsigned int t = (n) > 512 ? 512 : (n);

	unsigned int bt = (unsigned int)((n) / t);


	dim3 g(t);

	dim3 b(bt);



	bit_reverse_copy << <g, b >> > (A_d, n, B_d);



	long int m = 2;

	for (int i = 1; i <= log2((double)n); i++) {

		unsigned int x = ((n / 2) < 512) ? (n / 2) : 512;

		unsigned int bx = ((n / 2) / x);


		dim3 grid(x);

		dim3 blocks(bx);

		fft << <grid, blocks >> > (B_d, m);

		m *= 2;

	}

	hipEventRecord(stp2);

	hipMemcpy(A, B_d, size, hipMemcpyDeviceToHost);



	hipFree(A_d);

	hipFree(B_d);



	hipEventRecord(stp);



	hipEventSynchronize(stp);

	hipEventSynchronize(stp2);



	float milliseconds = 0;

	float milliseconds2 = 0;

	hipEventElapsedTime(&milliseconds, stt, stp);

	hipEventElapsedTime(&milliseconds2, stt2, stp2);



	cout << milliseconds << " milliseconds elapsed! (With copy)" << endl;

	cout << milliseconds2 << " milliseconds elapsed! (Without copy)" << endl;





	FILE* out;

	out = fopen("teste_cuda.txt", "a+");

	if (out) {

		fprintf(out, "%lf  -  %lf\n", milliseconds, milliseconds2);

		fclose(out);

	}

	free(A);



	cout << "The end..." << endl;

	int d;

	cin >> d;

}